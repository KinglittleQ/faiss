#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/impl/IcmEncoder.cuh>

#include <faiss/gpu/GpuResources.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/L2Norm.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/DeviceTensor.cuh>
#include <faiss/gpu/utils/MatrixMult.cuh>
#include <faiss/gpu/utils/Pair.cuh>
#include <faiss/gpu/utils/Reductions.cuh>

#include <hiprand/hiprand_kernel.h>

namespace faiss {
namespace gpu {

/** encode using iterative conditional mode
 *
 * For every subcode ci (i = 1, ..., M) of a vector, we fix the other
 * subcodes cj (j != i) and then find the optimal value of ci such
 * that minimizing the objective function.
 *
 * @param uterm  precomputed unary terms, size (n, M, K)
 * @param bterm  precomputed binary terms, size (M1, M2, K1, K2)
 * @param codes  output vector encodings, size (n, M)
 * @param M      number of codebooks
 * @param m      identify which subcode to condition on
 * @param K      number of codewords in a codebook
 */
template <int K>
__global__ void runIcmEncodeStep(
        const float* uterm,
        const float* bterm,
        int32_t* codes,
        int M,
        int m) {
    using KVPair = Pair<float, int>;
    constexpr int smemSize = (K + kWarpSize - 1) / kWarpSize;

    int id = blockIdx.x;
    int code = threadIdx.x;
    __shared__ KVPair smem[smemSize];

    KVPair obj(0.0f, code);
    obj.k = uterm[id * K + code];

    // unrolling this loop does not improve speed
    for (int m2 = 0; m2 < M; m2++) {
        if (m2 == m) {
            continue;
        }
        int32_t code2 = codes[id * M + m2];
        obj.k += bterm[m2 * K * K + code * K + code2];
    }

    __syncthreads();
    obj = blockReduceAll<KVPair, Min<KVPair>, false, false>(
            obj, Min<KVPair>(), smem);

    if (code == 0) {
        codes[id * M + m] = obj.v;
    }
}

template <int K>
__global__ void runEvaluate(
        const float* x,
        const float* codebooks,
        const int32_t* codes,
        float* obj, // output
        int n,
        int M,
        int dims) {
    int id = blockIdx.x; // index of the vector
    int d = threadIdx.x; // dimension
    extern __shared__ float smem[];

    float acc = 0.0f;

    // TODO: unroll M ?
    for (int m = 0; m < M; m++) {
        int32_t code = codes[id * M + m];
        acc += codebooks[m * K * dims + code * dims + d];
    }

    acc -= x[id * dims + d];
    acc = acc * acc;

    __syncthreads();
    acc = blockReduceAllSum<float, false, false>(acc, smem);

    if (d == 0) {
        obj[id] = acc;
    }
}

template <int K>
__global__ void runPerturbCodes(
        int seed,
        int32_t* codes,
        int n,
        int M,
        int nperts) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // index of the vector

    if (id >= n) {
        return;
    }

    // we have to initialize the state
    hiprandState_t state;
    hiprand_init(seed, id, 0, &state);

    for (int i = 0; i < nperts; i++) {
        int pos = int(hiprand_uniform(&state) * M);
        int32_t val = int32_t(hiprand_uniform(&state) * K);
        codes[id * M + pos] = val;
    }
}

__global__ void runSelectBest(
        int32_t* bestCodes,
        float* bestObjs,
        const int32_t* codes,
        const float* objs,
        int n,
        int M) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // index of the vector

    if (id >= n || objs[id] >= bestObjs[id]) {
        return;
    }

    bestObjs[id] = objs[id];
    for (int m = 0; m < M; m++) {
        bestCodes[id * M + m] = codes[id * M + m];
    }
}

__global__ void runNormAdd(float* bterm, const float* norm, int K) {
    int id = blockIdx.x;
    int code = threadIdx.x;

    bterm[id * K + code] += norm[code];
}

void IcmEncoderImpl::computeUnaryTerms(
        float* uterm,           // output, [M, n, K]
        const float* x,         // [n, d]
        const float* codebooks, // [M, K, d]
        int n,
        int dims) const {
    auto stream = res->getDefaultStreamCurrentDevice();
    auto handle = res->getBlasHandleCurrentDevice();

    DeviceTensor<float, 2, true> vecs(const_cast<float*>(x), {n, dims});
    for (int m = 0; m < M; m++) {
        auto cPtr = const_cast<float*>(codebooks + m * K * dims);
        auto bPtr = uterm + m * n * K;
        DeviceTensor<float, 2, true> ci(cPtr, {K, dims});
        DeviceTensor<float, 2, true> bi(bPtr, {n, K});
        runMatrixMult(
                bi,
                false,
                vecs,
                false,
                ci,
                true,
                -2.0f,
                0.0f,
                handle,
                stream);
    }

    DeviceTensor<float, 2, true> c(
            const_cast<float*>(codebooks), {M * K, dims});
    DeviceTensor<float, 1, true> norm(
            res.get(), makeTempAlloc(AllocType::Other, stream), {M * K});
    runL2Norm(c, true, norm, true, stream);

    for (int m = 0; m < M; m++) {
        auto bPtr = uterm + m * n * K;
        auto nPtr = norm.data() + m * K;
        runNormAdd<<<n, K, 0, stream>>>(bPtr, nPtr, K);
    }
}

void IcmEncoderImpl::computeBinaryTerms(
        float* bterm,  // output, [M, M, K, K]
        const float* codebooks,  // [M, K, d]
        int dims) const {

    auto stream = res->getDefaultStreamCurrentDevice();
    auto handle = res->getBlasHandleCurrentDevice();

    for (int m1 = 0; m1 < M; m1++) {
        for (int m2 = 0; m2 < M; m2++) {
            auto ptr1 = const_cast<float*>(codebooks + m1 * K * dims);
            auto ptr2 = const_cast<float*>(codebooks + m2 * K * dims);
            auto ptr3 = bterm + m1 * M * K * K + m2 * K * K;
            DeviceTensor<float, 2, true> c1(ptr1, {K, dims});
            DeviceTensor<float, 2, true> c2(ptr2, {K, dims});
            DeviceTensor<float, 2, true> b(ptr3, {K, K});
            runMatrixMult(
                    b,
                    false,
                    c1,
                    false,
                    c2,
                    true,
                    2.0f,
                    0.0f,
                    handle,
                    stream);
        }
    }
}


IcmEncoderImpl::IcmEncoderImpl(
        int M,
        int K,
        GpuResourcesProvider* prov,
        int device)
        : M(M), K(K), prov(prov), device(device) {
    res = prov->getResources();
}

void IcmEncoderImpl::setBinaryTerm(const float* codebooksHost, int dims) {
    DeviceScope scope(device);
    auto device = getCurrentDevice();
    auto stream = res->getDefaultStreamCurrentDevice();

    codebooks = toDeviceNonTemporary<float, 3>(
            res.get(),
            device,
            const_cast<float*>(codebooksHost),
            stream,
            {M, K, dims});
    // bterm = toDeviceNonTemporary<float, 4>(
    //         res.get(),
    //         device,
    //         const_cast<float*>(binaries),
    //         stream,
    //         {M, M, K, K});
    bterm = DeviceTensor<float, 4, true>(
            res.get(), makeDevAlloc(AllocType::Other, stream), {M, M, K, K});
    computeBinaryTerms(bterm.data(), codebooks.data(), dims);
}

template <int K>
void IcmEncoderImpl::encodeImpl(
        const float* xHost,
        const float* codebooksHost,
        int32_t* codesHost,
        std::mt19937& gen,
        int n,
        int dims,
        int nperts,
        int ilsIters,
        int icmIters) const {
    DeviceScope scope(device);
    auto device = getCurrentDevice();
    auto stream = res->getDefaultStreamCurrentDevice();

    auto codes = toDeviceTemporary<int32_t, 2>(
            res.get(), device, const_cast<int32_t*>(codesHost), stream, {n, M});
    auto x = toDeviceTemporary<float, 2>(
            res.get(), device, const_cast<float*>(xHost), stream, {n, dims});
    // auto codebooks = toDeviceTemporary<float, 3>(
    //         res.get(),
    //         device,
    //         const_cast<float*>(codebooksHost),
    //         stream,
    //         {M, K, dims});

    DeviceTensor<float, 3, true> uterm(
            res.get(), makeTempAlloc(AllocType::Other, stream), {M, n, K});
    computeUnaryTerms(uterm.data(), x.data(), codebooks.data(), n, dims);

    DeviceTensor<int32_t, 2, true> bestCodes(
            res.get(), makeTempAlloc(AllocType::Other, stream), {n, M});
    fromDevice<int32_t, 2>(codes, bestCodes.data(), stream);

    DeviceTensor<float, 1, true> bestObjs(
            res.get(), makeTempAlloc(AllocType::Other, stream), {n});

    DeviceTensor<float, 1, true> objs(
            res.get(), makeTempAlloc(AllocType::Other, stream), {n});

    const int smem = sizeof(float) * (dims + kWarpSize - 1) / kWarpSize;
    runEvaluate<K><<<n, dims, smem, stream>>>(
            x.data(),
            codebooks.data(),
            codes.data(),
            bestObjs.data(),
            n,
            M,
            dims);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    for (int i = 0; i < ilsIters; i++) {
        runPerturbCodes<K><<<numBlocks, blockSize, 0, stream>>>(
                gen(), codes.data(), n, M, nperts);

        for (int j = 0; j < icmIters; j++) {
            for (int m = 0; m < M; m++) {
                runIcmEncodeStep<K><<<n, K, 0, stream>>>(
                        uterm[m].data(), bterm[m].data(), codes.data(), M, m);
            }
        }

        runEvaluate<K><<<n, dims, smem, stream>>>(
                x.data(),
                codebooks.data(),
                codes.data(),
                objs.data(),
                n,
                M,
                dims);

        runSelectBest<<<numBlocks, blockSize, 0, stream>>>(
                bestCodes.data(),
                bestObjs.data(),
                codes.data(),
                objs.data(),
                n,
                M);

        codes.copyFrom(bestCodes, stream);
    }

    // copy back to host memory
    fromDevice<int32_t, 2>(bestCodes, codesHost, stream);
}

void IcmEncoderImpl::encode(
        const float* x,
        const float* codebooks,
        int32_t* codes,
        std::mt19937& gen,
        int n,
        int dims,
        int nperts,
        int ilsIters,
        int icmIters) const {
    FAISS_THROW_IF_NOT(K <= (1 << 16));

#define DISPATCH_K(nbits)         \
    case (1 << nbits):            \
        encodeImpl<(1 << nbits)>( \
                x,                \
                codebooks,        \
                codes,            \
                gen,              \
                n,                \
                dims,             \
                nperts,           \
                ilsIters,         \
                icmIters);        \
        break

    switch (K) {
        DISPATCH_K(1);
        DISPATCH_K(2);
        DISPATCH_K(3);
        DISPATCH_K(4);
        DISPATCH_K(5);
        DISPATCH_K(6);
        DISPATCH_K(7);
        DISPATCH_K(8); // K = 256
        DISPATCH_K(9);
        DISPATCH_K(10);
        DISPATCH_K(11);
        DISPATCH_K(12);
        DISPATCH_K(13);
        DISPATCH_K(14);
        DISPATCH_K(15);
        DISPATCH_K(16);
        default:
            FAISS_THROW_MSG("Invalid codebook size");
    }
#undef DISPATCH_K
}

} // namespace gpu
} // namespace faiss
