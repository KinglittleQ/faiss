#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/impl/IcmEncoder.cuh>

#include <faiss/gpu/GpuResources.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/L2Norm.cuh>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/DeviceTensor.cuh>
#include <faiss/gpu/utils/MatrixMult.cuh>
#include <faiss/gpu/utils/Pair.cuh>
#include <faiss/gpu/utils/Reductions.cuh>

#include <hiprand/hiprand_kernel.h>

namespace faiss {
namespace gpu {

extern __shared__ char smem[];

/** encode using iterative conditional mode
 *
 * For every subcode ci (i = 1, ..., M) of a vector, we fix the other
 * subcodes cj (j != i) and then find the optimal value of ci such
 * that minimizing the objective function.
 *
 * @param uterm  precomputed unary terms, size (n, M, K)
 * @param bterm  precomputed binary terms, size (M1, M2, K1, K2)
 * @param codes  output vector encodings, size (n, M)
 * @param M      number of codebooks
 * @param K      number of codewords in a codebook
 * @param m      identify which subcode to condition on
 */
__global__ void runIcmEncodeStep(
        const float* uterm,
        const float* bterm,
        int32_t* codes,
        int M,
        int K,
        int m) {
    using KVPair = Pair<float, int>;
    // constexpr int smemSize = (K + kWarpSize - 1) / kWarpSize;

    int id = blockIdx.x;
    int code = threadIdx.x;
    // __shared__ KVPair smem[smemSize];

    KVPair obj(0.0f, code);
    obj.k = uterm[id * K + code];

    // unrolling this loop does not improve speed
    for (int m2 = 0; m2 < M; m2++) {
        if (m2 == m) {
            continue;
        }
        int32_t code2 = codes[id * M + m2];
        obj.k += bterm[m2 * K * K + code * K + code2];
    }

    __syncthreads();
    obj = blockReduceAll<KVPair, Min<KVPair>, false, false>(
            obj, Min<KVPair>(), (KVPair*)smem);

    if (code == 0) {
        codes[id * M + m] = obj.v;
    }
}

__global__ void runEvaluate(
        const float* x,
        const float* codebooks,
        const int32_t* codes,
        float* obj, // output
        int n,
        int M,
        int K,
        int dims) {
    int id = blockIdx.x; // index of the vector
    int d = threadIdx.x; // dimension
    // extern __shared__ float smem[];

    float acc = 0.0f;

    // TODO: unroll M ?
    for (int m = 0; m < M; m++) {
        int32_t code = codes[id * M + m];
        acc += codebooks[m * K * dims + code * dims + d];
    }

    acc -= x[id * dims + d];
    acc = acc * acc;

    __syncthreads();
    acc = blockReduceAllSum<float, false, false>(acc, (float*)smem);

    if (d == 0) {
        obj[id] = acc;
    }
}

__global__ void runPerturbCodes(
        int seed,
        int32_t* codes,
        int n,
        int M,
        int K,
        int nperts) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // index of the vector

    if (id >= n) {
        return;
    }

    // we have to initialize the state
    hiprandState_t state;
    hiprand_init(seed, id, 0, &state);

    for (int i = 0; i < nperts; i++) {
        int pos = int(hiprand_uniform(&state) * M);
        int32_t val = int32_t(hiprand_uniform(&state) * K);
        codes[id * M + pos] = val;
    }
}

__global__ void runSelectBest(
        int32_t* bestCodes,
        float* bestObjs,
        const int32_t* codes,
        const float* objs,
        int n,
        int M) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // index of the vector

    if (id >= n || objs[id] >= bestObjs[id]) {
        return;
    }

    bestObjs[id] = objs[id];
    for (int m = 0; m < M; m++) {
        bestCodes[id * M + m] = codes[id * M + m];
    }
}

__global__ void runNormAdd(float* bterm, const float* norm, int K) {
    int id = blockIdx.x;
    int code = threadIdx.x;

    bterm[id * K + code] += norm[code];
}

IcmEncoderImpl::IcmEncoderImpl(
        int M,
        int K,
        int dims,
        GpuResourcesProvider* prov,
        int device)
        : M(M), K(K), dims(dims), prov(prov), device(device) {
    res = prov->getResources();
}

void IcmEncoderImpl::computeUnaryTerms(
        float* uterm,           // output, [M, n, K]
        const float* x,         // [n, d]
        const float* codebooks, // [M, K, d]
        int n) const {
    auto stream = res->getDefaultStreamCurrentDevice();
    auto handle = res->getBlasHandleCurrentDevice();

    DeviceTensor<float, 2, true> vecs(const_cast<float*>(x), {n, dims});
    for (int m = 0; m < M; m++) {
        auto cPtr = const_cast<float*>(codebooks + m * K * dims);
        auto bPtr = uterm + m * n * K;
        DeviceTensor<float, 2, true> ci(cPtr, {K, dims});
        DeviceTensor<float, 2, true> bi(bPtr, {n, K});
        runMatrixMult(
                bi, false, vecs, false, ci, true, -2.0f, 0.0f, handle, stream);
    }

    DeviceTensor<float, 2, true> c(
            const_cast<float*>(codebooks), {M * K, dims});
    DeviceTensor<float, 1, true> norm(
            res.get(), makeTempAlloc(AllocType::Other, stream), {M * K});
    runL2Norm(c, true, norm, true, stream);

    for (int m = 0; m < M; m++) {
        auto bPtr = uterm + m * n * K;
        auto nPtr = norm.data() + m * K;
        runNormAdd<<<n, K, 0, stream>>>(bPtr, nPtr, K);
    }
}

void IcmEncoderImpl::computeBinaryTerms(float* bterm, const float* codebooks)
        const {
    auto stream = res->getDefaultStreamCurrentDevice();
    auto handle = res->getBlasHandleCurrentDevice();

    for (int m1 = 0; m1 < M; m1++) {
        for (int m2 = 0; m2 < M; m2++) {
            auto ptr1 = const_cast<float*>(codebooks + m1 * K * dims);
            auto ptr2 = const_cast<float*>(codebooks + m2 * K * dims);
            auto ptr3 = bterm + m1 * M * K * K + m2 * K * K;
            DeviceTensor<float, 2, true> c1(ptr1, {K, dims});
            DeviceTensor<float, 2, true> c2(ptr2, {K, dims});
            DeviceTensor<float, 2, true> b(ptr3, {K, K});
            runMatrixMult(
                    b, false, c1, false, c2, true, 2.0f, 0.0f, handle, stream);
        }
    }
}


void IcmEncoderImpl::setBinaryTerm(const float* codebooksHost) {
    DeviceScope scope(device);
    auto device = getCurrentDevice();
    auto stream = res->getDefaultStreamCurrentDevice();

    codebooks = toDeviceNonTemporary<float, 3>(
            res.get(),
            device,
            const_cast<float*>(codebooksHost),
            stream,
            {M, K, dims});
    bterm = DeviceTensor<float, 4, true>(
            res.get(), makeDevAlloc(AllocType::Other, stream), {M, M, K, K});
    computeBinaryTerms(bterm.data(), codebooks.data());
}

void IcmEncoderImpl::encodeImpl(
        int32_t* codesHost,
        const float* xHost,
        const float* codebooksHost,
        std::mt19937& gen,
        int n,
        int K,
        int nperts,
        int ilsIters,
        int icmIters) const {
    DeviceScope scope(device);
    auto device = getCurrentDevice();
    auto stream = res->getDefaultStreamCurrentDevice();

    auto codes = toDeviceTemporary<int32_t, 2>(
            res.get(), device, const_cast<int32_t*>(codesHost), stream, {n, M});
    auto x = toDeviceTemporary<float, 2>(
            res.get(), device, const_cast<float*>(xHost), stream, {n, dims});

    DeviceTensor<float, 3, true> uterm(
            res.get(), makeTempAlloc(AllocType::Other, stream), {M, n, K});
    computeUnaryTerms(uterm.data(), x.data(), codebooks.data(), n);

    DeviceTensor<int32_t, 2, true> bestCodes(
            res.get(), makeTempAlloc(AllocType::Other, stream), {n, M});
    fromDevice<int32_t, 2>(codes, bestCodes.data(), stream);

    DeviceTensor<float, 1, true> bestObjs(
            res.get(), makeTempAlloc(AllocType::Other, stream), {n});

    DeviceTensor<float, 1, true> objs(
            res.get(), makeTempAlloc(AllocType::Other, stream), {n});

    const int evaluateSmem = sizeof(float) * (dims + kWarpSize - 1) / kWarpSize;
    const int encodeSmem =
            sizeof(Pair<float, int>) * (K + kWarpSize - 1) / kWarpSize;

    runEvaluate<<<n, dims, evaluateSmem, stream>>>(
            x.data(),
            codebooks.data(),
            codes.data(),
            bestObjs.data(),
            n,
            M,
            K,
            dims);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    for (int i = 0; i < ilsIters; i++) {
        runPerturbCodes<<<numBlocks, blockSize, 0, stream>>>(
                gen(), codes.data(), n, M, K, nperts);

        for (int j = 0; j < icmIters; j++) {
            for (int m = 0; m < M; m++) {
                runIcmEncodeStep<<<n, K, encodeSmem, stream>>>(
                        uterm[m].data(),
                        bterm[m].data(),
                        codes.data(),
                        M,
                        K,
                        m);
            }
        }

        runEvaluate<<<n, dims, evaluateSmem, stream>>>(
                x.data(),
                codebooks.data(),
                codes.data(),
                objs.data(),
                n,
                M,
                K,
                dims);

        runSelectBest<<<numBlocks, blockSize, 0, stream>>>(
                bestCodes.data(),
                bestObjs.data(),
                codes.data(),
                objs.data(),
                n,
                M);

        codes.copyFrom(bestCodes, stream);
    }

    // copy back to host memory
    fromDevice<int32_t, 2>(bestCodes, codesHost, stream);
}

void IcmEncoderImpl::encode(
        int32_t* codes,
        const float* x,
        const float* codebooks,
        std::mt19937& gen,
        int n,
        int nperts,
        int ilsIters,
        int icmIters) const {
    FAISS_THROW_IF_NOT(K <= (1 << 16));

    encodeImpl(
            codes, x, codebooks, gen, n, K, nperts, ilsIters, icmIters);
}

} // namespace gpu
} // namespace faiss
